#include "hip/hip_runtime.h"
#ifdef fail
        #!/bin/bash
        # NOTE you can chmod 0755 this file and then execute it to compile (or just copy and paste)
        gcc -o hashblock hashblock.c -lssl
        exit 0
#endif
 
//#include <openssl/sha.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <sys/time.h>
#include "sha256.cu"
#include "strtol.cu"
#include "strlen.cu"

#define SHA256_DIGEST_SIZE 64
#define NUM_BLOCKS 1024

// this is the block header, it is 80 bytes long (steal this code)
typedef struct block_header {
        unsigned int    version;
        // dont let the "char" fool you, this is binary data not the human readable version
        unsigned char   prev_block[32];
        unsigned char   merkle_root[32];
        unsigned int    timestamp;
        unsigned int    bits;
        unsigned int    nonce;
} block_header;
 
double When()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1e-6);
}

// we need a helper function to convert hex to binary, this function is unsafe and slow, but very readable (write something better)
__device__ void hex2bin(unsigned char* dest, const char* src)
{
        int c, pos;
        char buf[3];
 
        pos=0;
        c=0;
        buf[2] = 0;
        while(c < strlen2(src))
        {
                // read in 2 characaters at a time
                buf[0] = src[c++];
                buf[1] = src[c++];
                // convert them to a interger and recast to a char (uint8)
                dest[pos++] = (unsigned char)strtol2(buf, NULL, 16);
        }
       
}
 
// this function is mostly useless in a real implementation, were only using it for demonstration purposes
__device__ void print_hash(unsigned char hash[])
{
   int idx;
   for (idx=0; idx < 32; idx++)
      printf("%02x",hash[idx]);
   printf("\n");
}
 
// this function swaps the byte ordering of binary data, this code is slow and bloated (write your own)
__device__ void byte_swap(unsigned char* data) {
        int c;
        unsigned char tmp[SHA256_DIGEST_SIZE];
       
        c=0;
        while(c<SHA256_DIGEST_SIZE)
        {
                tmp[c] = data[SHA256_DIGEST_SIZE-(c+1)];
                c++;
        }
       
        c=0;
        while(c<SHA256_DIGEST_SIZE)
        {
                data[c] = tmp[c];
                c++;
        }
}

__global__ void doCalc( unsigned int *seed) {
    block_header header;
    // we need a place to store the checksums
    unsigned char hash1[32];
    unsigned char hash2[32];
    SHA256_CTX sha256_pass1, sha256_pass2;
   
    header.version =        2;
    hex2bin(header.prev_block,              "000000000000000117c80378b8da0e33559b5997f2ad55e2f7d18ec1975b9717");
    hex2bin(header.merkle_root,             "871714dcbae6c8193a2bb9b2a69fe1c0440399f38d94b3a0f1b447275a29978a");
    header.timestamp =      1392872245;
    header.bits =           419520339;
    header.nonce =          0;

    byte_swap(header.prev_block);
    byte_swap(header.merkle_root);

    header.nonce = (*seed *  blockDim.x * NUM_BLOCKS) + blockIdx.x * blockDim.x + threadIdx.x;
    if(threadIdx.x == 0) printf("nonce: %d\n", header.nonce);
    sha256_init(&sha256_pass1);
    // then you 'can' feed data to it in chuncks, but here were just making one pass cause the data is so small
    sha256_update(&sha256_pass1, (unsigned char*)&header, sizeof(block_header));
    // this ends the sha256 session and writes the checksum to hash1
    sha256_final(&sha256_pass1,hash1);
       
    //second hash
    sha256_init(&sha256_pass2);
    sha256_update(&sha256_pass2, hash1, SHA256_DIGEST_SIZE);
    sha256_final(&sha256_pass2, hash2);
    if ( header.nonce == 0 || header.nonce == 3 || header.nonce == 856192328 ) {
        //hexdump((unsigned char*)&header, sizeof(block_header));
        printf("%u:\n", header.nonce);
        byte_swap(hash2);
        printf("Target Second Pass Checksum: \n");
        print_hash(hash2);
    }
}

int main() {
    int blocksize = 4;
    int threads = 4;

    int hashes = 0;
    int results = 0;
    int counter = 0;
    unsigned int *d_counter;
    hipMalloc((void**)&d_counter, sizeof(d_counter));
    
    double start = When();
    double timer = When() - start;
    while ( timer < 10.0){
        hipMemcpy(d_counter, &counter, sizeof(counter), hipMemcpyHostToDevice);
        doCalc<<< blocksize, threads >>>( d_counter);
        hashes += blocksize*threads;
        counter++;
        timer = When() - start;
        //printf("%d iterations\n",counter);
        hipDeviceSynchronize();
  //      hipMemcpy(&results , d_counter, sizeof(counter), hipMemcpyDeviceToHost);
  //      printf("counter: %d \n",counter); 
    }

    printf("number of hashs per second = %f\n",hashes / (When() - start) );

 
    return 0;
}